#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello() {
    printf("Hello World\n");
}

int main() {
    cuda_hello<<<1,1>>>();
    hipError_t err = hipDeviceSynchronize();

    if (err != hipSuccess)
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        
    return 0;
}
