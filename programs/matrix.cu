#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void multiplication_kernel(float* A, float* B, float* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockIdx.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    size_t size = N * N * sizeof(float);

    float *h_A = new float[N*N];
    float *h_B = new float[N*N];
    float *h_C = new float[N*N];

    for (int i = 0; i < N*N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N+15)/16, (N+15)/16);

    while (true) {
        multiplication_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
        hipDeviceSynchronize();
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    delete[] h_A; delete[] h_B, delete[] h_C;

    return 0;
}
