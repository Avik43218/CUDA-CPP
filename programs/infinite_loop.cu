
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void infinite_loop(long long int msg) {
    
    while (true) {
        printf("%d", msg);
        msg += 2;
        printf("\n");
    }
}

int main() {

    long long int a = 1;

    infinite_loop<<<1024,1024>>>(a);
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
        printf("CUDA Error: %s\n", hipGetErrorString(err));

    hipDeviceSynchronize();

    return 0;
}
